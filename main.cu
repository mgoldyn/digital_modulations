#include "hip/hip_runtime.h"
#include "inc\qpsk.h"
#include "inc\bpsk.h"
#include "inc\types.h"
#include "inc\consts.h"
#include "inc\psk_common.h"
#include "inc\amp_mod.h"
#include "inc\freq_mod.h"
#include "inc\bpsk_cuda.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
extern "C"
{
C_DELLEXPORT float* psk_cos_lut;
C_DELLEXPORT float* modulated_data;
C_DELLEXPORT float* dynamic_data;

C_DELLEXPORT int32_t init_func(float amplitude,
                               float freq,
                               int32_t cos_factor_idx,
                               int32_t n_bits,
                               int32_t* bit_stream,
                               char* mod)
{
    char bps[] = "bpsk";
    char bpsc[] = "bpskc";
    char qps[] = "qpsk";
    char qpsc[] = "qpskc";
    char am[]  = "am";
    char amc[]  = "amc";
    char fm[]  = "fm";
    char fmc[]  = "fmc";
    
    if(!strcmp(mod, bps))
    {
        const psk_params params = {amplitude, freq, cos_factor_idx};
        int32_t n_cos_samples   = get_n_cos_samples(params.cos_factor_idx);
        clock_t t; t = clock();
        psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
        modulated_data = (float*)malloc(sizeof(float) * n_cos_samples * n_bits);
        if(!psk_cos_lut || !modulated_data)
        {
            return 1;
        }

        init_psk_cos_lut(&params, psk_cos_lut);
        
        modulate_bpsk(n_cos_samples, n_bits, bit_stream, psk_cos_lut, modulated_data);
        t = clock() - t;
        double time_taken = ((double)t)/CLOCKS_PER_SEC;
        printf("mod C = %s, time = %.30lf\n", mod, time_taken);
    }
    else if(!strcmp(mod, qps))
    {
        const psk_params params = {amplitude, freq, cos_factor_idx};
        int32_t n_cos_samples   = get_n_cos_samples(params.cos_factor_idx);

        psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
        modulated_data = (float*)malloc(sizeof(float) * n_cos_samples * n_bits / 2);
        if(!psk_cos_lut || !modulated_data)
        {
            return 1;
        }

        init_psk_cos_lut(&params, psk_cos_lut);
        modulate_qpsk(n_cos_samples, n_bits, bit_stream, psk_cos_lut, modulated_data);
        printf("mod C = %s\n", mod);
    }
    else if(!strcmp(mod, qpsc))
    {
        const psk_params params = {amplitude, freq, cos_factor_idx};
        int32_t n_cos_samples   = get_n_cos_samples(params.cos_factor_idx);

        psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
        modulated_data = (float*)malloc(sizeof(float) * n_cos_samples * n_bits / 2);
        if(!psk_cos_lut || !modulated_data)
        {
            return 1;
        }

        init_psk_cos_lut(&params, psk_cos_lut);
        modulate_qpsk_cuda(n_cos_samples, n_bits, bit_stream, psk_cos_lut, modulated_data);
        printf("mod C = %s\n", mod);
    }
    else if(!strcmp(mod, am))
    {
        const psk_params params = {amplitude, freq, cos_factor_idx};
        int32_t n_cos_samples   = get_n_cos_samples(params.cos_factor_idx);

        psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
        modulated_data = (float*)malloc(sizeof(float) * n_cos_samples * n_bits);
        if(!psk_cos_lut || !modulated_data)
        {
            return 1;
        }

        init_psk_cos_lut(&params, psk_cos_lut);
        modulate_am(n_cos_samples, n_bits, bit_stream, psk_cos_lut, modulated_data);
        printf("mod C = %s\n", mod);
    }
    else if(!strcmp(mod, amc))
    {
        const psk_params params = {amplitude, freq, cos_factor_idx};
        int32_t n_cos_samples   = get_n_cos_samples(params.cos_factor_idx);

        psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
        modulated_data = (float*)malloc(sizeof(float) * n_cos_samples * n_bits);
        if(!psk_cos_lut || !modulated_data)
        {
            return 1;
        }

        init_psk_cos_lut(&params, psk_cos_lut);
        modulate_am_cuda(n_cos_samples, n_bits, bit_stream, psk_cos_lut, modulated_data);
        printf("mod C = %s\n", mod);
    }
    else if(!strcmp(mod, fm))
    {
        const psk_params params = {amplitude, freq, cos_factor_idx};
        int32_t n_cos_samples   = get_n_cos_samples(params.cos_factor_idx);

        psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
        modulated_data = (float*)malloc(sizeof(float) * n_cos_samples * n_bits);
        if(!psk_cos_lut || !modulated_data)
        {
            return 1;
        }

        init_fm_cos_lut(&params, psk_cos_lut);
        modulate_fm(n_cos_samples, n_bits, bit_stream, psk_cos_lut, modulated_data);
        printf("mod C = %s\n", mod);
    }
    else if(!strcmp(mod, fmc))
    {
        const psk_params params = {amplitude, freq, cos_factor_idx};
        int32_t n_cos_samples   = get_n_cos_samples(params.cos_factor_idx);

        psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
        modulated_data = (float*)malloc(sizeof(float) * n_cos_samples * n_bits);
        if(!psk_cos_lut || !modulated_data)
        {
            return 1;
        }

        init_fm_cos_lut(&params, psk_cos_lut);
        modulate_fm_cuda(n_cos_samples, n_bits, bit_stream, psk_cos_lut, modulated_data);
        printf("mod C = %s\n", mod);
    }
    else if(!strcmp(mod, bpsc))
    {
        hipFree(0);
        const psk_params params = {amplitude, freq, cos_factor_idx};
        int32_t n_cos_samples   = get_n_cos_samples(params.cos_factor_idx);

        psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
        modulated_data = (float*)malloc(sizeof(float) * n_cos_samples * n_bits);
        if(!psk_cos_lut || !modulated_data)
        {
            return 1;
        }

        init_psk_cos_lut(&params, psk_cos_lut);
        clock_t t = clock();
        modulate_bpsk_cuda(n_cos_samples, n_bits, bit_stream, psk_cos_lut, modulated_data);
        t = clock() - t;
        double time_taken = ((double)t)/CLOCKS_PER_SEC;
        printf("mod C = %s, time = %.30lf\n", mod, time_taken);
        
    }
    else
    {
        return 1;
    }
    
    return 0;
}

C_DELLEXPORT void memory_free()
{
   free(dynamic_data);
   free(psk_cos_lut);
   free(modulated_data);
}
}

int main(void)
{
    // printf("dupa");
    // int32_t bit_stream[] = {0, 1, 1, 0, 0, 0, 1, 1};
    // psk_params params = {1, 5, 2};
    // int32_t n_cos_samples = get_n_cos_samples(params.cos_factor_idx);
    // psk_cos_lut    = (float*)malloc(sizeof(float) * n_cos_samples * N_SIGNAL_PERIODS);
    // modulated_data =  (float*)malloc(sizeof(float) * n_cos_samples * 8);
    // init_psk_cos_lut(&params, psk_cos_lut);
    // modulate_bpsk_cuda(n_cos_samples, 8, bit_stream, psk_cos_lut, modulated_data);
    // printf("dupa");
    // int32_t i = 0;

    int32_t bit_stream[] = {0,1,0,1};//,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1};
    char bps[] = "fmc";
    hipFree(0);
    init_func(1,
        1,
        2,
        4,
        bit_stream,
        bps);
         int i = 0;
             for(; i < 360; ++i)
     {
         printf("mod[%d] = %f \n", i, modulated_data[i]);
     }
    memory_free();
//    int32_t bit_stream[] = {0,1,0,1};//,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1,0,1};
    char bps2[] = "fm";
    hipFree(0);
    init_func(1,
              1,
              2,
              4,
              bit_stream,
              bps2);
    i = 0;
    for(; i < 360; ++i)
    {
        printf("mod[%d] = %f \n", i, modulated_data[i]);
    }

    memory_free();
    return 0;
}
