#include "hip/hip_runtime.h"
#include "..\inc\qpsk.h"
#include "..\inc\consts.h"
#include "..\inc\psk_common.h"
#include "..\inc\cuda_common.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define QPSK_PHASE_01  45
#define QPSK_PHASE_11 135
#define QPSK_PHASE_10 225
#define QPSK_PHASE_00 315

#define N_CUDA_ELEM 128

void modulate_qpsk(int32_t n_cos_samples,
                   int32_t n_bits,
                   const int32_t*  bit_stream,
                   const float*  signal_data,
                   float*  modulated_signal)
{
    int32_t bit_idx = 0, data_idx = 0;
    int32_t phase_shift;

    for(; bit_idx < n_bits; bit_idx += 2, ++data_idx)
    {
        if(!bit_stream[bit_idx])
        {
            if(!bit_stream[bit_idx + 1])
            {
                phase_shift = QPSK_PHASE_00;
            }
            else
            {
                phase_shift = QPSK_PHASE_01;
            }
        }
        else
        {
            if(!bit_stream[bit_idx + 1])
            {
                phase_shift = QPSK_PHASE_10;
            }
            else
            {
                phase_shift = QPSK_PHASE_11;
            }
        }
        
        set_phase_shift(n_cos_samples, phase_shift, signal_data, &modulated_signal[data_idx * n_cos_samples]);
    }
}

__global__ void
set_phase_offset_qpsk_cuda(const int32_t* bit_stream,
                           int32_t n_bits,
                           int32_t* phase_offset)
{
    int32_t bit_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(bit_idx < n_bits)
    {
        if(!bit_stream[bit_idx * 2])
        {
            if(!bit_stream[(bit_idx * 2) + 1])
            {
                phase_offset[bit_idx] = QPSK_PHASE_00;
            }
            else
            {
                phase_offset[bit_idx] = QPSK_PHASE_01;
            }
        }
        else
        {
            if(!bit_stream[(bit_idx * 2) + 1])
            {
                phase_offset[bit_idx] = QPSK_PHASE_10;
            }
            else
            {
                phase_offset[bit_idx] = QPSK_PHASE_11;
            }
        }
    }
}

__global__ void
set_phase_shift_qspk_cuda(int32_t* phase_offset,
                          int32_t n_cos_samples,
                          int32_t n_bits,
                          const float* signal_data,
                          float* modulated_signal)
{

    int32_t bit_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(bit_idx < n_bits)
    {
        float* modulated_signal_ptr = &modulated_signal[bit_idx * n_cos_samples];
        int32_t sig_idx = 0;
        for(; sig_idx < n_cos_samples; ++sig_idx)
        {
            modulated_signal_ptr[sig_idx] = signal_data[phase_offset[bit_idx] + sig_idx];
        }
    }
}

void modulate_qpsk_cuda(int32_t n_cos_samples,
                        int32_t n_bits,
                        const int32_t*  bit_stream,
                        const float*  signal_data,
                        float*  modulated_signal)
{
    float* d_modulated_signal = get_modulated_signal();
    float* d_signal_data = get_signal_data();
    int32_t* d_bit_stream = get_bit_stream();
    int32_t* d_phase_offset;
    int32_t n_elem = n_bits / 2 < N_CUDA_ELEM ? n_bits / 2 : N_CUDA_ELEM;

    hipMalloc((void**)&d_phase_offset, sizeof(int32_t) * n_bits / 2);
    hipMemcpy(d_bit_stream, bit_stream, sizeof(int32_t) * n_bits, hipMemcpyHostToDevice);
    hipMemcpy(d_signal_data, signal_data, sizeof(float) * n_cos_samples * 2, hipMemcpyHostToDevice);

    int threadsPerBlock = 16;
    int blocksPerGrid   = ( N_CUDA_ELEM + threadsPerBlock - 1) / threadsPerBlock;

    int32_t n_data = n_bits / 2;
    int32_t data_idx = 0;
    for(; data_idx < n_data; data_idx += N_CUDA_ELEM)
    {
        int32_t n_cuda_bits = n_data < N_CUDA_ELEM ? n_data : data_idx + N_CUDA_ELEM > n_data ? n_data - data_idx: N_CUDA_ELEM;
        set_phase_offset_qpsk_cuda<<<blocksPerGrid, threadsPerBlock>>>(&d_bit_stream[data_idx],
                                                                       n_cuda_bits,
                                                                       &d_phase_offset[data_idx]);
    }


    for(data_idx = 0; data_idx < n_data; data_idx += N_CUDA_ELEM)
    {
        int32_t n_cuda_bits = n_data < N_CUDA_ELEM ? n_data : data_idx + N_CUDA_ELEM > n_data ? n_data - data_idx : N_CUDA_ELEM;
        set_phase_shift_qspk_cuda<<<blocksPerGrid, threadsPerBlock>>>(&d_phase_offset[data_idx],
                                                                      n_cos_samples,
                                                                      n_cuda_bits,
                                                                      d_signal_data,
                                                                      d_modulated_signal);
        hipMemcpy(&modulated_signal[data_idx * n_cos_samples],
                   d_modulated_signal,
                   sizeof(float) * n_cos_samples * n_cuda_bits,
                   hipMemcpyDeviceToHost);
    }

    hipFree((void*)d_phase_offset);
}
