#include "hip/hip_runtime.h"
#include "..\inc\bpsk_cuda.h"
#include "..\inc\consts.h"
#include "..\inc\psk_common.h"
#include <hip/hip_runtime.h>
#include <cstring>
#include <stdio.h>

#define BSPK_PHASE_0 90
#define BPSK_PHASE_1 270

#define N_CUDA_ELEM 128

__global__ void
set_phase_offset_cuda(const int32_t* bit_stream,
                      int32_t n_bits,
                      int32_t* phase_offset)
{
    int32_t bit_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(bit_idx < n_bits)
    {
        if(!bit_stream[bit_idx])
        {
            phase_offset[bit_idx] = (int32_t)((float)BSPK_PHASE_0 * ((float)360)/ N_MAX_DEGREE);
        }
        else
        {
            phase_offset[bit_idx] = (int32_t)((float)BPSK_PHASE_1 * ((float)360)/ N_MAX_DEGREE);
        }
    }
}

__global__ void
set_phase_shift_cuda(int32_t* phase_offset,
                     int32_t n_cos_samples,
                     int32_t n_bits,
                     const float* signal_data,
                     float* modulated_signal)
{

    int32_t bit_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(bit_idx < n_bits)
    {
        float* modulated_signal_ptr = &modulated_signal[bit_idx * n_cos_samples];
        int32_t sig_idx = 0;
        for(; sig_idx < n_cos_samples; ++sig_idx)
        {
            modulated_signal_ptr[sig_idx] = signal_data[phase_offset[bit_idx] + sig_idx];
        }
//        modulated_signal[sig_idx] = signal_data[scaled_phase_shift + sig_idx];
    }
}

void modulate_bpsk_cuda(int32_t n_cos_samples,
                        int32_t n_bits,
                        const int32_t* bit_stream,
                        const float* signal_data,
                        float* modulated_signal)
{
    float* d_modulated_signal;
    float* d_signal_data;
    int32_t* d_bit_stream;
    int32_t* d_phase_offset;
    int32_t n_elem = n_bits < N_CUDA_ELEM ? n_bits : N_CUDA_ELEM;

    hipMalloc((void**)&d_modulated_signal, sizeof(float) * n_cos_samples * n_elem);
    hipMalloc((void**)&d_signal_data, sizeof(float) * n_cos_samples * 2);
    hipMalloc((void**)&d_bit_stream, sizeof(int32_t) * n_bits);
    hipMalloc((void**)&d_phase_offset, sizeof(int32_t) * n_bits);
    hipMemcpy(d_bit_stream, bit_stream, sizeof(int32_t) * n_bits, hipMemcpyHostToDevice);
    hipMemcpy(d_signal_data, signal_data, sizeof(float) * n_cos_samples * 2, hipMemcpyHostToDevice);

    int threadsPerBlock = 16;
    int blocksPerGrid = (N_CUDA_ELEM + threadsPerBlock - 1) / threadsPerBlock;

    int32_t bit_idx = 0;
    for(; bit_idx < n_bits; bit_idx += N_CUDA_ELEM)
    {
        int32_t n_cuda_bits = n_bits < N_CUDA_ELEM ? n_bits : bit_idx + N_CUDA_ELEM > n_bits ? n_bits - bit_idx: N_CUDA_ELEM;
        set_phase_offset_cuda<<<blocksPerGrid, threadsPerBlock>>>(d_bit_stream,
                                                                  n_cuda_bits,
                                                                  &d_phase_offset[bit_idx]);
    }
    for(bit_idx = 0; bit_idx < n_bits; bit_idx += N_CUDA_ELEM)
    {
        int32_t n_cuda_bits = n_bits < N_CUDA_ELEM ? n_bits : bit_idx + N_CUDA_ELEM > n_bits ? n_bits - bit_idx: N_CUDA_ELEM;
        set_phase_shift_cuda<<<blocksPerGrid, threadsPerBlock>>>(&d_phase_offset[bit_idx],
                                                                 n_cos_samples,
                                                                 n_cuda_bits,
                                                                 d_signal_data,
                                                                 d_modulated_signal);
        hipMemcpy(&modulated_signal[bit_idx * n_cos_samples],
                   d_modulated_signal,
                   sizeof(float) * n_cos_samples * n_cuda_bits,
                   hipMemcpyDeviceToHost);
    }
    hipFree((void*)d_modulated_signal);
    hipFree((void*)d_signal_data);
    hipFree((void*)d_bit_stream);
    hipFree((void*)d_phase_offset);

}