#include "hip/hip_runtime.h"
#include "..\inc\bpsk_cuda.h"
#include "..\inc\consts.h"
#include "..\inc\psk_common.h"
#include "..\inc\cuda_common.h"
#include <hip/hip_runtime.h>
#include <cstring>
#include <stdio.h>

#define BSPK_PHASE_0 90
#define BPSK_PHASE_1 270

#define N_CUDA_ELEM 512

__global__ void
set_phase_shift_cuda(int32_t* bit_stream,
                     int32_t n_cos_samples,
                     int32_t n_bits,
                     const float* signal_data,
                     float* modulated_signal)
{

    int32_t bit_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(bit_idx < n_bits)
    {
        int32_t phase_offset;
        if(!bit_stream[bit_idx])
        {
            phase_offset = (int32_t)((float)BSPK_PHASE_0 * ((float)360)/ N_MAX_DEGREE);
        }
        else
        {
            phase_offset = (int32_t)((float)BPSK_PHASE_1 * ((float)360)/ N_MAX_DEGREE);
        }
        float* modulated_signal_ptr = &modulated_signal[bit_idx * n_cos_samples];
        int32_t sig_idx = 0;
        for(; sig_idx < n_cos_samples; ++sig_idx)
        {
            modulated_signal_ptr[sig_idx] = signal_data[phase_offset + sig_idx];
        }
    }
}

void modulate_bpsk_cuda(int32_t n_cos_samples,
                        int32_t n_bits,
                        const int32_t* bit_stream,
                        const float* signal_data,
                        float* modulated_signal)
{
    float* d_modulated_signal = get_modulated_signal();
    float* d_signal_data = get_signal_data();
    int32_t* d_bit_stream = get_bit_stream();
    int32_t n_elem = n_bits < N_CUDA_ELEM ? n_bits : N_CUDA_ELEM;

//    hipMalloc((void**)&d_modulated_signal, sizeof(float) * n_cos_samples * n_bits);
//    hipMalloc((void**)&d_signal_data, sizeof(float) * n_cos_samples * 2);
//    hipMalloc((void**)&d_bit_stream, sizeof(int32_t) * n_bits);
//    hipMemcpy(d_bit_stream, bit_stream, sizeof(int32_t) * n_bits, hipMemcpyHostToDevice);
    hipMemcpy(d_signal_data, signal_data, sizeof(float) * n_cos_samples * 2, hipMemcpyHostToDevice);

    int threadsPerBlock = 16;
    int blocksPerGrid = (N_CUDA_ELEM + threadsPerBlock - 1) / threadsPerBlock;
    hipStream_t prolog_stream;
    hipStream_t main_stream[8];
    hipStream_t epilog_stream;
    hipStreamCreate(&prolog_stream);
    for(int32_t i  = 0; i < (n_bits - N_CUDA_ELEM) / N_CUDA_ELEM; ++i)
    {
        hipStreamCreate(&main_stream[i]);
    }
//    prolog
    int32_t bit_idx = 0;
    int32_t n_cuda_prolog_bits = n_bits < N_CUDA_ELEM ? n_bits : bit_idx + N_CUDA_ELEM > n_bits ? n_bits - bit_idx: N_CUDA_ELEM;
    hipMemcpyAsync(&d_bit_stream[bit_idx],
                    bit_stream,
                    sizeof(int32_t) * n_cuda_prolog_bits,
                    hipMemcpyHostToDevice,
                    prolog_stream);

    set_phase_shift_cuda<<<blocksPerGrid, threadsPerBlock, 0, prolog_stream>>>(&d_bit_stream[bit_idx],
                                                                               n_cos_samples,
                                                                               n_cuda_prolog_bits,
                                                                               d_signal_data,
                                                                               &d_modulated_signal[bit_idx * n_cos_samples]);
    hipMemcpyAsync(&modulated_signal[bit_idx * n_cos_samples],
                    &d_modulated_signal[bit_idx * n_cos_samples],
                    sizeof(float) * n_cos_samples * n_cuda_prolog_bits,
                    hipMemcpyDeviceToHost,
                    prolog_stream);
//  main loop
    int32_t n_main_loop_bits = n_bits - N_CUDA_ELEM;
    bit_idx = N_CUDA_ELEM;
    for(; bit_idx < n_main_loop_bits; bit_idx += N_CUDA_ELEM)
    {
        hipMemcpyAsync(&d_bit_stream[bit_idx],
                        bit_stream,
                        sizeof(int32_t) * N_CUDA_ELEM,
                        hipMemcpyHostToDevice,
                        main_stream[(bit_idx / N_CUDA_ELEM) - 1]);

        set_phase_shift_cuda<<<blocksPerGrid, threadsPerBlock, 0, main_stream[(bit_idx / N_CUDA_ELEM) - 1]>>>(&d_bit_stream[bit_idx],
                                                                                                               n_cos_samples,
                                                                                                               N_CUDA_ELEM,
                                                                                                               d_signal_data,
                                                                                                               &d_modulated_signal[bit_idx * n_cos_samples]);
        hipMemcpyAsync(&modulated_signal[bit_idx * n_cos_samples],
                        &d_modulated_signal[bit_idx * n_cos_samples],
                        sizeof(float) * n_cos_samples * N_CUDA_ELEM,
                        hipMemcpyDeviceToHost,
                        main_stream[(bit_idx / N_CUDA_ELEM) - 1]);
    }

//    epilog
//printf("mgoldyn bit_idx = %d\n", bit_idx);
    int32_t n_epilog_bits = n_bits - bit_idx;
    if(n_epilog_bits > 0)
    {
        hipStreamCreate(&epilog_stream);
        hipMemcpyAsync(&d_bit_stream[bit_idx],
                        bit_stream,
                        sizeof(int32_t) * n_epilog_bits,
                        hipMemcpyHostToDevice,
                        epilog_stream);

        set_phase_shift_cuda<<<blocksPerGrid, threadsPerBlock, 0, epilog_stream>>>(&d_bit_stream[bit_idx],
                                                                                   n_cos_samples,
                                                                                   n_epilog_bits,
                                                                                   d_signal_data,
                                                                                   &d_modulated_signal[bit_idx * n_cos_samples]);
        hipMemcpyAsync(&modulated_signal[bit_idx * n_cos_samples],
                        &d_modulated_signal[bit_idx * n_cos_samples],
                        sizeof(float) * n_cos_samples * n_epilog_bits,
                        hipMemcpyDeviceToHost,
                        epilog_stream);
        hipStreamDestroy(epilog_stream);
    }
    for(int32_t i  = 0; i < (n_bits - N_CUDA_ELEM) / N_CUDA_ELEM; ++i)
    {
        hipStreamDestroy(main_stream[i]);
    }
    hipStreamDestroy(prolog_stream);

//    hipFree((void*)d_modulated_signal);
//    hipFree((void*)d_signal_data);
//    hipFree((void*)d_bit_stream);

}